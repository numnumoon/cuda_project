#include <hip/hip_runtime.h>

__global__ void add(int *a, int *b, int *c, int N) {
    int i = threadIdx.x;
    if (i < N) {
        c[i] = a[i] + b[i];
    }
}

void launchAddKernel(int *d_a, int *d_b, int *d_c, int N) {
    add<<<1, N>>>(d_a, d_b, d_c, N);
    hipDeviceSynchronize();
}
